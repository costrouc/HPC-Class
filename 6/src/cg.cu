#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "timing.cpp"


#define RTOLERANCE     10e-16
#define ATOLERANCE     10e-16

//=============================================================================
// Standard CG routine in double precision arithmetic
//=============================================================================

// Reference SpMV product on the PCU
void dsmv(double *h_A, int *h_I, int *h_J, int N, double *h_X, double *h_Y)
{
   double res;
   for(int i=0; i<N; i++)
   {
      res=0;

        for(int j=h_I[i];j<h_I[i+1];j++)
        {
           res+=h_A[j]*h_X[h_J[j]];
        }
      h_Y[i]=res;
   }
}


void CGd(int dofs, int & num_of_iter,  double *x, double *b,
         double *h_A, int *h_I, int *h_J,
         double rtol = RTOLERANCE ){
  double *r=new double[dofs], *d=new double[dofs], *z=new double[dofs];
  double r0, den, nom, nom0, betanom, alpha, beta;
  int i, j;

  nom = 0.0;
  for(j=0; j<dofs; j++){
    x[j] = 0.;
    r[j] = d[j] = b[j];

    nom += r[j]*r[j];
  }
  nom0 = nom;                                 // nom = r dot r
  dsmv(h_A, h_I, h_J, dofs, r, z);            //   z = A r
  den = 0.0;

  for(j=0; j<dofs; j++)
    den += z[j]*r[j];                         // den = z dot r

  if ( (r0 = nom * rtol) < ATOLERANCE) r0 = ATOLERANCE;
  if (nom < r0)
    return;

  if (den <= 0.0) {
    printf("Operator A is not postive definite. (Ar,r) = %f\n", den);
    return;
  }

  // printf("Iteration : %4d  Norm: %f\n", 0, nom);

  // start iteration
  for(i= 1; i<num_of_iter ;i++) {
    alpha = nom/den;

    betanom = 0.0;
    for(j=0;j<dofs; j++){
      x[j] += alpha*d[j];                         //  x = x + alpha d
      r[j] -= alpha*z[j];                         //  r = r - alpha z
      betanom += r[j]*r[j];                       //  betanom = r dot r
    }

    // printf("Iteration : %4d  Norm: %f\n", i, betanom);
    if ( betanom < r0 ) {
      num_of_iter = i;
      break;
    }

    beta = betanom/nom;                           // beta = betanom/nom
    for(j=0;j<dofs; j++)
      d[j] = r[j] + beta * d[j];                  // d = r + beta d           

    dsmv(h_A, h_I, h_J, dofs, d, z);              // z = A d
    den = 0.;
    for(j=0;j<dofs; j++)
      den += d[j]*z[j];                           // den = d dot z
    nom = betanom;
  } // end iteration

  printf( "      (r_0, r_0) = %e\n", nom0);
  printf( "      (r_N, r_N) = %e\n", betanom);
  printf( "      Number of CG iterations: %d\n", i);

  if (rtol == RTOLERANCE) {
    dsmv(h_A, h_I, h_J, dofs, x, r);              //    r = A x
    den = 0.0;
    for(j=0; j<dofs; j++){
      r[j] = b[j] - r[j];                         //    r = b  - r
      den += r[j]*r[j];
    }
    printf( "      || r_N ||   = %f\n", sqrt(den));
  }

  delete [] r;
  delete [] z;
  delete [] d;
}


//=============================================================================
// Standard CG routine in double precision arithmetic on the GPU
//=============================================================================

// SpMV on the GPU
#define num_threads 32
__global__ void dsmv_kernel(double* A, int *I, int *J, int n, double *d_X, double *d_Y)
{
   int ind = blockIdx.x*num_threads + threadIdx.x;

   if (ind < n){
      I += ind;
 
      int j, last=I[1];
      double res = 0.f;

      for(j=I[0];j<last;j++)
         res += A[j] * d_X[ J[j] ];

      d_Y[ind] = res;
   }
}


void dsmv_gpu(double *d_A, int *d_I, int *d_J, int N, double *d_X, double *d_Y)
{
   dim3 grid(N/num_threads, 1, 1);
   dim3 threads(num_threads, 1, 1);

   dsmv_kernel<<<grid, threads>>>(d_A, d_I, d_J, N, d_X, d_Y);
}


void CGd_GPU(int dofs, int & num_of_iter,  double *x, double *b,
             double *d_A, int *d_I, int *d_J, double *dwork,
             double rtol = RTOLERANCE ){

  double *r = dwork;
  double *d = dwork + dofs;
  double *z = dwork + 2*dofs;

  double r0, den, nom, nom0, betanom, alpha, beta;
  int i;

  hipblasDscal(dofs, 0.f, x, 1);        // x = 0
  hipblasDcopy(dofs, b, 1, r, 1);       // r = b
  hipblasDcopy(dofs, b, 1, d, 1);       // d = b
  nom = hipblasDnrm2(dofs, r, 1);       // nom = || r ||
  nom = nom * nom;

  nom0 = nom;                          // nom = r dot r
  dsmv_gpu(d_A, d_I, d_J, dofs, r, z); // z = A r
  den = hipblasDdot(dofs, z, 1, r, 1);  // den = z dot r

  if ( (r0 = nom * rtol) < ATOLERANCE) r0 = ATOLERANCE;
  if (nom < r0)
    return;

  if (den <= 0.0) {
    printf("Operator A is not postive definite. (Ar,r) = %f\n", den);
    return;
  }

  // printf("Iteration : %4d  Norm: %f\n", 0, nom);

  // start iteration
  for(i= 1; i<num_of_iter ;i++) {
    alpha = nom/den;
    hipblasDaxpy(dofs,  alpha, d, 1, x, 1);         // x = x + alpha d
    hipblasDaxpy(dofs, -alpha, z, 1, r, 1);         // r = r - alpha z
    betanom = hipblasDnrm2(dofs, r, 1);             // betanom = || r ||
    betanom = betanom * betanom;                   // betanom = r dot r

    // printf("Iteration : %4d  Norm: %f\n", i, betanom);
    if ( betanom < r0 ) {
      num_of_iter = i;
      break;
    }

    beta = betanom/nom;                           // beta = betanom/nom
    hipblasDscal(dofs, beta, d, 1);                // d = beta*d
    hipblasDaxpy(dofs, 1.f, r, 1, d, 1);           // d = d + r
    dsmv_gpu(d_A, d_I, d_J, dofs, d, z);          // z = A d
    den = hipblasDdot(dofs, d, 1, z, 1);           // den = d dot z

    nom = betanom;
  } // end iteration

  printf( "      (r_0, r_0) = %e\n", nom0);
  printf( "      (r_N, r_N) = %e\n", betanom);
  printf( "      Number of CG iterations: %d\n", i);

  if (rtol == RTOLERANCE) {
    dsmv_gpu(d_A, d_I, d_J, dofs, x, r);          // r = A x
    hipblasDaxpy(dofs,  -1.f, b, 1, r, 1);         // r = r - b
    den = hipblasDnrm2(dofs, r, 1);                // den = || r ||
    printf( "      || r_N ||   = %f\n", den);
  }
}
//============================================================================

int main(int argc,char **argv)
{

   hipInit( 0 );
   cublasInit( );

   TimeStruct start, end;

   int N, i, NNZ, inc=0, filelines=0;
   int read,col1,row1;
   float val1;
   FILE   *DataFile;

   //======================Reading file=======================================
   //==========================================================================

   printf("\n....... Reading matrix.output ......................... \n");

   if ((DataFile = fopen("matrix.output", "r")) == NULL)
      printf("\nCan't read matrix.output\n");

   fscanf(DataFile,"%d%d%d", &N, &N, &NNZ);

   int current_col = 0, k = 0, *nnz_row;
   nnz_row   = (int*)malloc( sizeof(int)*(N+1));
   nnz_row[k] = inc;
   //=======================Memory allocation=================
   //===========================================================

   double *h_Y, *h_X, *d_X, *d_Y, *h_Y1, *dwork;
   double *h_A,*d_A;
   int *h_J, *h_I, *d_J, *d_I;

   h_X=(double*)malloc(N*sizeof(double)); 
   if (h_X==NULL) printf("fail to allocate h_X\n"), exit(1);
  
   h_A=(double*)malloc((NNZ+1)*sizeof(double));
   if (h_A==NULL) printf("fail to allocate h_A\n"), exit(1);
    
   h_Y=(double*)malloc(N*sizeof(double));
   if (h_Y==NULL) printf("fail to allocate h_Y\n"), exit(1);
    
   h_Y1=(double*)malloc(N*sizeof(double));
   if (h_Y1==NULL) printf("fail to allocate h_Y1\n"), exit(1);
    
   h_A=(double*)malloc(NNZ*sizeof(double)); 
   if (h_A==NULL) printf("fail to allocate h_A\n"), exit(1);
    
   h_J=(int*)malloc((NNZ+1)*sizeof(int));    
   if (h_J==NULL) printf("fail to allocate h_J\n"), exit(1);
    
   h_I=(int*)malloc((N+1)*sizeof(int));  
   if (h_I==NULL) printf("fail to allocate h_I\n"), exit(1);

   for(i=0; i<N; i++)
     h_X[i] = 1.f*rand()/RAND_MAX;

   for(i=0;i<NNZ;i++){
      read=fscanf(DataFile,"%d%d%f",&col1,&row1,&val1);

      if(read!=3)break;
      h_J[filelines]=col1-1;

      if (current_col == row1-1)
        nnz_row[k]++;
      else
      {
        current_col = row1-1;
        k++;
        inc=1;
        nnz_row[k] = 1;
      }

      h_A[filelines]=val1;

      filelines++;
   }
   printf("file lines: %d\n", filelines);

   fclose(DataFile);
   fprintf(stderr,"File is closed\n");
 
   h_I[0]=0;
   i = 0;
   for(i=1;i<=N;i++)
     h_I[i]=h_I[i-1] + nnz_row[i-1];

   printf("N = %d\n", N);
   //===========================================================
   //=============sparse Matrix vector product on CPU================
   //==============================================================
   start = get_current_time();
   dsmv(h_A, h_I, h_J, N, h_X, h_Y);          // h_Y = h_A * h_X
   end = get_current_time();

   printf("CPU Processing time: %f (ms) \n", GetTimerValue(start,end));
   printf("Speed: %f GFlops \n", 2.*NNZ/
           (1.*1000000*GetTimerValue(start,end)));

   //======================================================
   //=====================GPU=============================
   //===================================================

   printf("....... allocating GPU memory ........................... \n\n");

   hipMalloc((void**)&dwork,3*N*sizeof(double));
   hipMalloc((void**)&d_X,N*sizeof(double));
   hipMalloc((void**)&d_Y,N*sizeof(double));
   hipMalloc((void**)&d_A,(NNZ+1)*sizeof(double));
   hipMalloc((void**)&d_I,(N+1)*sizeof(int));
   hipMalloc((void**)&d_J,(NNZ+1)*sizeof(int));

   hipMemcpy(d_A, h_A, NNZ*sizeof(double),    hipMemcpyHostToDevice);
   hipMemcpy(d_J, h_J, (NNZ+1)*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(d_I, h_I, (N+1)*sizeof(int),  hipMemcpyHostToDevice);
   hipMemcpy(d_X, h_X, N*sizeof(double),      hipMemcpyHostToDevice);

   printf("memory allocated\n");

   start = get_current_time();
   dsmv_gpu(d_A, d_I, d_J, N, d_X, d_Y);
   end = get_current_time();

   hipMemcpy(h_Y1, d_Y, N*sizeof(double), hipMemcpyDeviceToHost);

   //==========================================================================
   //=======print the result( first three values )from GPU=====================
   printf("\n....................................................... \n");
   double norm = 0.f;
   for(i=0;i<N;i++)
     norm += (h_Y[i] - h_Y1[i])*(h_Y[i] - h_Y1[i]);

   //==========================================================================
   printf("GPU Processing time: %f (ms) \n", GetTimerValue(start,end));
   printf("Speed: %f GFlops \n", 2.*NNZ/
           (1.*1000000*GetTimerValue(start, end)));
   printf("|| Y_GPU - Y_CPU ||_2 = %f \n", sqrt(norm));

   //==========================================================================
   // Solve  h_A * h_X = h_Y on the CPU using CG
   int max_num_iters = 5000;
   printf("\n....... Solving Ax = b using CG on the CPU ............ \n");

   start = get_current_time();
   CGd( N, max_num_iters, h_X, h_Y,
        h_A, h_I, h_J);
   end = get_current_time();

   printf("Time (s) = %f\n", GetTimerValue(start,end)/1000.);

   //==========================================================================
   // Solve  d_A * d_X = d_Y on the GPU using CG
   printf("\n....... Solving Ax = b using CG on the GPU ............ \n");

   start = get_current_time();
   CGd_GPU(N, max_num_iters, d_X, d_Y, d_A, d_I, d_J, dwork);
   end = get_current_time();

   printf("Time (s) = %f\n\n", GetTimerValue(start,end)/1000.);

   hipFree(d_X);
   hipFree(d_Y);
   hipFree(d_A);
   hipFree(d_I);
   hipFree(d_J);
   hipFree(dwork);

   free(h_A);
   free(h_X);
   free(h_Y);
   free(h_Y1);
   free(h_J);
   free(h_I);
}
